#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <assert.h>
#include <iostream>
#include <random>
#include <hipblas.h>

#include "linear_layer.hh"
#include "nn_exception.hh"

#define LEARNING_RATE 0.01

__global__ void ReluActivationForward(float* Z, float* A,float* Stored_Z, int Z_x_dim, int Z_y_dim) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < Z_x_dim * Z_y_dim) {
		A[index] = fmaxf(Z[index], 0);
                Stored_Z[index] = A[index];
	}
}


__global__ void ReluActivationBackprop(float* Z, float* dA, float* dZ, int Z_x_dim, int Z_y_dim) {
	
	//int nnodes = 2708;
	//int num_test_nodes = nnodes - (0.6*nnodes);
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < Z_x_dim * Z_y_dim) {
		if (Z[index] > 0) {
			dZ[index] = dA[index];
		}
		else {
			dZ[index] = 0;
		}
		//Adding it to quickly see if I can set output of node agg 0 for test nodes
		/*
		if(index < num_test_nodes) {
			dZ[index] = 0;
		}*/
	}
	/*
	if((row > 2700)) {
		printf("ReLU x = %d, y = %d, dZ = %f, dA = %f\n", row, i, dZ[i + dA_y_dim * row], dA[i + dA_y_dim * row]); 
	}*/
}
__global__ void linearLayerForward( float* W, float* A, float* Z, float* b,
                                                                           int W_x_dim, int W_y_dim,
                                                                           int A_x_dim, int A_y_dim){
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
  
    int Z_x_dim = A_x_dim;
    int Z_y_dim = W_x_dim;
  
    float Z_value = 0;
  
    if( row < Z_x_dim && col < Z_y_dim){
       for(int i=0; i< W_y_dim; i=i+1){
           Z_value += W[i + W_y_dim * col] * A[i + A_y_dim * row]; 
       }
       Z[row * Z_y_dim + col] = Z_value + b[col]; 
      // if(Z[row * Z_y_dim + col] > 0)
      //    printf("Z[%d]: %f\n", row * Z_y_dim + col, Z[row * Z_y_dim + col]);
    }
}

__global__ void linearLayerForwardAddBias( float* Z, float* bias, int numFeatures) {

    // APARNA TODO: fuse bias addition and reLU application
    // APARNA TODO: if this takes a lot of time -- can merge computations for some features like fuseGNN
    //Add Z: #nodes * #labels , b: labels * 1 (or 1 * labels) doesn't matter
  
    //APARNA TODO: maybe doing an inner loop where we process > 1 node per CTA will help  -- will reduce launch overhead

    /*
    for(int feature = threadIdx.x ; feature < numFeatures; feature += blockDim.x) {
	Z[blockIdx.x * numFeatures + feature] = Z[blockIdx.x * numFeatures + feature] + bias[feature];
    }*/
	
    Z[blockIdx.x * numFeatures + threadIdx.x] = Z[blockIdx.x * numFeatures + threadIdx.x] + bias[threadIdx.x];
    
}

__global__ void linearLayerBackprop( float* W, float* dZ, float*dA,
                                                                    int W_x_dim, int W_y_dim,
                                                                    int dZ_x_dim, int dZ_y_dim){
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
  
    int dA_x_dim = dZ_x_dim;
    int dA_y_dim = W_y_dim;
  
    float dA_value = 0.0f;
  	if (row < dA_x_dim && col < dA_y_dim) {
		    for (int i = 0; i < W_x_dim; i++) {
			      dA_value += -1 * W[i * W_y_dim + col] * dZ[ i + dZ_y_dim * row];
		    }
		    dA[row * dA_y_dim + col] = dA_value;
	  }
}

__global__ void linearLayerUpdateWeights(  float* W, float* dW,
						int W_x_dim, int W_y_dim,
						float learning_rate) {

	//W = W - (n) * dW

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if( x < W_x_dim && y < W_y_dim) {
	    W[x * W_y_dim + y] += (-1) * (learning_rate) * dW[x * W_y_dim + y];
	}
}

/*
//Reduces mxn array into 1xm array
__global__ void reduce_array(volatile scalar_t* sdata, unsigned int tid, unsigned int reduce_len, unsigned int f_dim){

    __shared__ scalar_t s_feature[blockSize];


    while (reduce_len > 1){
        __syncthreads();
        // add the remainer
        if ((tid < f_dim) && (reduce_len % 2 == 1)){
            sdata[tid] += sdata[tid + f_dim * (reduce_len - 1)];
        }
        reduce_len /= 2;
        if (tid < f_dim * reduce_len){
            sdata[tid] += sdata[tid + f_dim * reduce_len];
        }
    }
}
*/

__global__ void linearLayerUpdateBias(  float* dZ, float* b,
										int dZ_x_dim, int dZ_y_dim,
										int b_x_dim,
										float learning_rate) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < dZ_x_dim * dZ_y_dim) {
		int dZ_x = index % dZ_y_dim;
		int dZ_y = index / dZ_y_dim;
		atomicAdd(&b[dZ_y], - learning_rate * (dZ[dZ_y * dZ_y_dim + dZ_x] / dZ_y_dim));
	}
}


//__global__ void linearLayerUpdateBias(  float* dZ, float* b,
//										int dZ_x_dim, int dZ_y_dim,
//										int b_x_dim,
//										float learning_rate) {
//	int index = blockIdx.x * blockDim.x + threadIdx.x;
//	int tid = threadIdx.x;
//
//	//Setting to a high value
//	extern __shared__ float buffer[];
//	
//	//Assuming #of output features > # of threads
//	if(tid < dZ_y_dim) {
//		buffer[tid] = 0;
//	}
//	__syncthreads();
//
//	if (index < dZ_x_dim * dZ_y_dim) {
//		int dZ_x = index / dZ_y_dim;
//		int dZ_y = index % dZ_y_dim;
//		atomicAdd(&buffer[dZ_y], dZ[dZ_x * dZ_y_dim + dZ_y]);
//	}
//
//	__syncthreads();
//
//	if(tid < dZ_y_dim) {
//		atomicAdd(&b[tid], -learning_rate*buffer[tid]/dZ_y_dim);
//	}
//}


void LinearLayer::runGEMM(Matrix& A, Matrix& B, Matrix& C, bool transposeA, bool transposeB) {
	//The take transpose function is for back propagation --> we multiply A.B' instead of A.B if this is turned on

	// Create a handle for CUBLAS
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	
	// Do the actual multiplication
	//alpha * op(A) * op(B) + beta * OP(C)
	// C(m,n) = A(m,k) * B(k,n)

	int m = C.shape.x;
	int n = C.shape.y;
	int k = transposeA ? B.shape.x : A.shape.y;

	//int lda=k,ldb=n,ldc=n;
	int lda=transposeA ? m : k;  //mxk
	//int ldb= n; //transposeB ? n : k;  //  kxn
	int ldb= transposeB ? k : n;  //  kxn
	int ldc=n; //mxn

	const float alf = 1;
	const float bet = 0;

	const float *alpha = &alf;
	const float *beta = &bet;

	//Note: This function can't support the case when both transposeA and B are set to 1	
	hipblasSgemm(handle, 
		    transposeB ? HIPBLAS_OP_T : HIPBLAS_OP_N, 
		    transposeA ? HIPBLAS_OP_T : HIPBLAS_OP_N, 
		    n, m, k, alpha, B.data_device, ldb, A.data_device, lda, beta, C.data_device, ldc);
	
	//print_kernel<<<1,1>>>(Z.data_device);

	// Destroy the handle
	hipblasDestroy(handle);
}

LinearLayer::LinearLayer(std::string name,int layer_num, Shape W_shape):
    W(W_shape),b(W_shape.y,1),dW(W_shape)
{
    this->name = name;
    this->layer_num = layer_num;
//    std::cout << "updated layer name\n";
    b.allocateCudaMemory();
//    std::cout << "b allocated\n";
    W.allocateMemory();
    dW.allocateMemory();
//    std::cout << "w allocated\n";
    initializeBiasWithZeros();
//   std::cout << "bias initialized\n";
    initializeWeightsRandomly();
//    std::cout << "weights initialized\n";
}

void LinearLayer::free_matrix(){
  dW.freeMem();
}

LinearLayer::~LinearLayer()
{ };

void LinearLayer::initializeWeightsRandomly(){
    std::default_random_engine generator;
    std::normal_distribution<float> normal_distribution(0.0, 0.1);
//    std::cout << "W.shape.x:" << W.shape.x <<"\n";	
//    std::cout << "W.shape.y:" << W.shape.y <<"\n";	
    for(int x = 0; x < W.shape.x; x++){
	for(int y = 0 ; y < W.shape.y; y++){
	     W[x * W.shape.y + y] = normal_distribution(generator)*0.1;	
	     //printf("W[%d] = %f\n", (x * W.shape.y + y), W[x * W.shape.y + y]);
	}
    }
//    std::cout << "copying data from host to device\n";
    W.copyHostToDevice();
    free(W.data_host);
}

void LinearLayer::initializeBiasWithZeros() {
	//for (int x = 0; x < b.shape.x; x++) {
	//	b[x] = 0;
	//}
	//b.copyHostToDevice();
        hipMemset(b.data_device, 0, b.shape.x * b.shape.y* sizeof(float));
}

__global__ void print_kernel_lin(float *A, int size, std::string str) {
	for(int i=0; i<size; i++) {
		if(A[i] != 0.0) {
		    printf("The value of %s[%d] = %f\n", str, i, A[i]);
		}
	}
}


Matrix& LinearLayer::forward(Matrix& A, bool training, bool freeMatrix){
//   std::cout << " Linear forward A.x:" << A.shape.x << "\n";
//  std::cout << " Linear forward A.y:" << A.shape.y << "\n";
//  std::cout << " Linear forward W.x:" << W.shape.x << "\n";
//  std::cout << " Linear forward W.y:" << W.shape.y << "\n";
//   std::cout << " Linear forward A address:" << A.data_device << "\n";
    assert(W.shape.x = A.shape.y);
   // std::cout << "Linear layer forward\n";
    //std::cout<< "Linear Layer ptr:" << A.data_device << "\n";
    this->A = A;

    //std::cout<< "Linear Layer ptr:" << A.data_device << "\n";
    Shape Z_shape(A.shape.x,W.shape.y);
    Z.allocateCuda(Z_shape);
    computeAndStoreLayerOutput(A);
    if(layer_num != 2){
	stored_Z.allocateCuda(Z.shape);
	dim3 block_size(256);
	dim3 num_of_blocks((Z.shape.y * Z.shape.x + block_size.x - 1) / block_size.x);
	ReluActivationForward<<<num_of_blocks, block_size>>>(Z.data_device, Z.data_device,stored_Z.data_device, Z.shape.x, Z.shape.y);
        
    }
//    std::cout << "Linear Layer forward\n";
    NNException::throwIfDeviceErrorOccurred("Cannot perform Linear Layer forward propagation");
    
//    std::cout << " Linear forward shape.x:" << Z.shape.x << "\n";
//    std::cout << " Linear forward shape.y:" << Z.shape.y << "\n";
//    std::cout << " Linear forward A shape.x:" << A.shape.x << "\n";
//    std::cout << " Linear forward A shape.y:" << A.shape.y << "\n";
//    std::cout << " Linear forward A address:" << A.data_device << "\n";
    if(training == false) {
	if(freeMatrix) {
            A.freeMem();
	}
     }

    return Z;
	
}


__global__ void print_weight_sum(float *W, float *dW, int size) {
	float w_sum = 0;
	float dw_sum = 0;

	float w_sum_mod = 0;
	float dw_sum_mod = 0;
	for(int i=0; i<size; i++) {
		w_sum += W[i];
		dw_sum += dW[i];
		w_sum_mod +=   (W[i] > 0) ?  W[i] :  -W[i]; 
		dw_sum_mod += (dW[i] > 0) ? dW[i] : -dW[i]; 
	}

	printf("The value of Weight Sum = %f, dW sum = %f\n", w_sum, dw_sum);
	printf("The value of MOD Weight Sum = %f, dW sum = %f\n", w_sum_mod, dw_sum_mod);
}

void LinearLayer::computeAndStoreLayerOutput(Matrix& A) {

	runGEMM(A, W, Z, false, false);	
	//Num CTAs = #nodes, #threads = min(256, numFeatures)
	int threadsPerBlock = std::min(256, (int) W.shape.y);
	linearLayerForwardAddBias<<<(Z.shape.x + threadsPerBlock - 1)/threadsPerBlock, threadsPerBlock>>>(Z.data_device, b.data_device, Z.shape.y);
	
}

Matrix& LinearLayer::backprop(Matrix& dZ, float learning_rate, bool freeMatrix) {
      //  std::cout << "Linear layer backword\n";
        if(layer_num != 2){
         	dim3 block_size(256);
         	dim3 num_of_blocks((stored_Z.shape.y * stored_Z.shape.x + block_size.x - 1) / block_size.x);
         	ReluActivationBackprop<<<num_of_blocks, block_size>>>(stored_Z.data_device, dZ.data_device,dZ.data_device, stored_Z.shape.x, stored_Z.shape.y);
                NNException::throwIfDeviceErrorOccurred("Cannot perform ReLU back propagation");

        } 

	dA.allocateCuda(A.shape);
	//dW.allocateCuda(W.shape); //A'.dZ

      //  std::cout << "Linear Layer backward\n";
	//print_kernel_lin<<<1,1>>>(dZ.data_device, dZ.shape.x*dZ.shape.y, "dZ - pre backprop ");
	computeAndStoreBackpropError(dZ);
	NNException::throwIfDeviceErrorOccurred("Cannot perform back propagation.");

	/*
	if(dZ.shape.y == 32) {
	    printf("Printing dZ of lin 1 layer\n");
	    print_kernel_lin<<<1,1>>>(dZ.data_device, dZ.shape.x*dZ.shape.y, "dZ - in backprop ");
	}*/
	updateBias(dZ, learning_rate);
	NNException::throwIfDeviceErrorOccurred("Cannot perform bias update.");
        
        //std::cout << " A ptr: " << A.data_device << "\n";
        //std::cout << " A last :" << A.data_device + (A.shape.x *  A.shape.y * 4) << "\n";  
        //std::cout << " dZ ptr: " << dZ.data_device << "\n";
        //std::cout << " dZ last :" << dZ.data_device + (dZ.shape.x *  dZ.shape.y * 4) << "\n";  
        //std::cout << " Linear backward shape dZ.x:" << dZ.shape.x << "\n";
        //std::cout << " Linear backward shape dZ.y:" << dZ.shape.y << "\n";
        //std::cout << " Linear backward shape A.x:" << A.shape.x << "\n";
        //std::cout << " Linear backward shape A.y:" << A.shape.y << "\n";
	updateWeights(dZ, learning_rate);
	NNException::throwIfDeviceErrorOccurred("Cannot perform weights update.");

        //std::cout << " Linear backward shape.x:" << dA.shape.x << "\n";
        //std::cout << " Linear backward shape.y:" << dA.shape.y << "\n";
        stored_Z.freeMem();
        //dZ.freeMem();
	//dW.freeMem();
        if(A.device_allocated == true){
            if(freeMatrix){
               A.freeMem();
            }
        }
	return dA;
}

void LinearLayer::computeAndStoreBackpropError(Matrix& dZ) {

	//std::cout << "dZ.x = " << dZ.shape.x << ", dZ.y = " << dZ.shape.y << std::endl;
	//std::cout << "dA.x = " << dA.shape.x << ", dA.y = " << dA.shape.y << std::endl;

	//W: 10x7, dz: 2708x7, dA: 2708x10 
	// So dA = dz.W'
	runGEMM(dZ, W, dA, false, true);	
	//TODO: need to multiply dA with -1. <<< Are we sure??? -- why not do that in dZ calculation?>>>

	/*
	if(dZ.shape.y == 7) {
		printf("Printing dA\n");
		print_kernel_lin<<<1,1>>>(dA.data_device, dA.shape.x*dA.shape.y, "dA ");
		hipDeviceSynchronize();
	}*/

}

void LinearLayer::updateWeights(Matrix& dZ, float learning_rate) {

	//dW = A'.dZ
	//dw: 10x7, A: 2708x10, dZ: 2708x7	
	runGEMM(A, dZ, dW, true, false);	

	//print_weight_sum<<<1,1>>>(W.data_device, dW.data_device, W.shape.x*W.shape.y);
	
	//Weight size is 1433x16 and 16x7	

	//W = W - (n) * dW
	dim3 block_size(16, 16);
	dim3 num_of_blocks((W.shape.x + block_size.x - 1) / block_size.x,(W.shape.y + block_size.y - 1) / block_size.y);
	linearLayerUpdateWeights<<<num_of_blocks, block_size>>>(W.data_device,
								dW.data_device,
								W.shape.x, W.shape.y,
								learning_rate);
}

void LinearLayer::updateBias(Matrix& dZ, float learning_rate) {

	//db: 1x7
	//The operation is dB = dZ.(reduce in Xdim) so 2708x7 --> 1x7 
	//Then b = b - (n) * dB
	
	//Need to write a reduction kernel for the first line
	//print_kernel_lin<<<1,1>>>(dZ.data_device, dZ.shape.x*dZ.shape.y, "dZ - pre bias ");


	dim3 block_size(512);
	dim3 num_of_blocks( (dZ.shape.y * dZ.shape.x + block_size.x - 1) / block_size.x);
	linearLayerUpdateBias<<<num_of_blocks, block_size, dZ.shape.y>>>(dZ.data_device,
							     b.data_device,
							     dZ.shape.x, dZ.shape.y,
							     b.shape.x, learning_rate);

	//printf("Bias X: %d, Y: %d\n", b.shape.x, b.shape.y);
	//print_kernel_lin<<<1,1>>>(b.data_device, b.shape.x*b.shape.y, "bias");
}

int LinearLayer::getXdim() const {
	return W.shape.x;
}

int LinearLayer::getYdim() const {
	return W.shape.y;
}

Matrix LinearLayer::getWeightsMatrix() const {
	return W;
}

Matrix LinearLayer::getBiasVector() const {
	return b;
}
	    
