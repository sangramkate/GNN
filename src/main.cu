#include <iostream>
#include <time.h>
#include <fstream>
#include <string>
#include <stdlib.h>
#include <sstream>

#include "NeuralNetwork.hh"
#include "linear_layer.hh"
#include "activation.hh"
#include "softmax.hh"
#include "nodeaggregator.hh"
#include "nn_exception.hh"
#include "costfunction.hh"
#include "csr_graph.h"
#include "data.hh" 



float computeAccuracy(const Matrix& predictions, const Matrix& targets, int *node_array, int num_test_nodes);

int main() {

        hipSetDevice(0);
        //std::fstream myfile("/net/ohm/export/iss/inputs/Learning/cora-labels.txt", std::ios_base::in);
        std::fstream label_info;
        label_info.open("datasets/cora/label_val.csv", std::ios::in);
        int* label = (int *) malloc(2708*7*sizeof(int));
        int i = 0;
        std::string line, word, temp;
        while(std::getline(label_info,line)) {
            std::stringstream s(line);
            while(std::getline(s,word,',')) {
                label[i] = stoi(word);
                i++;
            }

        }
        std::cout << i-1 << "\n";
	srand( time(NULL) );

	//CoordinatesDataset dataset(100, 21);
	CostFunction bce_cost;

//Code for extracting data from dataset files starts here
        CSRGraph graph;
        char gr_file[]="cora.gr";
        char binFile[]="cora-feat.bin";
        int nnodes = 0,nedges = 0;
        int feature_size = 1433;
        int label_size = 7;
        graph.read(gr_file,&nnodes,&nedges);
        int* d_row_start;
        int* d_edge_dst;
        float* d_edge_data;
        hipError_t alloc;
        int nnz = 5429;
        printf("2*nnz+2708 = %d\n",2*nnz+2708);
        int* h_row_start = (int*)malloc((2708+1) * sizeof(int));
        int* h_edge_dst = (int*)malloc((2*nnz+2708) * sizeof(int));
        alloc = hipMalloc(&d_row_start,(2708+1) * sizeof(int));
        if(alloc != hipSuccess) {
            printf("malloc for row info failed\n");
        }
        alloc = hipMalloc(&d_edge_dst,(2*nnz+2708) * sizeof(int));
        if(alloc != hipSuccess) {
            printf("malloc for col info failed\n");
        }
        float* d_B;

        float* h_B = (float *)malloc((2708) * feature_size * sizeof(float));
	if(h_B == NULL)
	    printf("h_B malloc failed\n");
        alloc = hipMalloc(&d_B, (2708) * feature_size * sizeof(float));
        if(alloc != hipSuccess) {
            printf("hipMalloc failed for features matrix\n");
        }
        alloc = hipMalloc(&d_edge_data,(2*nnz+2708) * sizeof(float));
        if(alloc != hipSuccess) {
            printf("malloc failed \n");
        }
	float* h_edge_data = (float *)malloc((2*nnz+2708) * sizeof(float));
        for(int i=0;i<(2*nnz+2708);i++)
            h_edge_data[i] = 1.0;
	alloc = hipMemcpy(d_edge_data, h_edge_data, ((2*nnz+2708) *sizeof(float)), hipMemcpyHostToDevice);
        if(alloc != hipSuccess) {
        printf("Feature matrix memcpy failed\n");
        }

//Filling up the sparse matrix info
        //graph.readFromGR(gr_file , binFile , d_row_start, d_edge_dst , d_B, feature_size);
        std::fstream feature_info;
        feature_info.open("datasets/cora/feature_val.csv", std::ios::in);
        i = 0;
        while(std::getline(feature_info,line)) {
            std::stringstream s(line);
            while(std::getline(s,word,',')) {
                h_B[i] = stof(word);
                i++;
            }

        }

        printf("node * feature = %d\n",i);
        std::fstream row_start_info;
        row_start_info.open("datasets/cora/row_start.csv", std::ios::in);
        i = 0;
        while(std::getline(row_start_info,line)) {
            std::stringstream s(line);
            while(std::getline(s,word,',')) {
                h_row_start[i] = stoi(word);
                i++;
            }

        }
        printf("node + 1  = %d\n",i);

        std::fstream edge_dst_info;
        edge_dst_info.open("datasets/cora/edge_dst.csv", std::ios::in);
        i = 0;
        while(std::getline(edge_dst_info,line)) {
            std::stringstream s(line);
            while(std::getline(s,word,',')) {
                h_edge_dst[i] = stoi(word);
                //printf("h_edge_dst[%d] = %d\n",i,h_edge_dst[i]);
                i++;
            }

        } 
        printf("egdges = %d\n",i);

        alloc = hipMemcpy(d_B, h_B, (2708 * 1433 *sizeof(float)), hipMemcpyHostToDevice);
        hipMemcpy(d_row_start, h_row_start,(2708+1) * sizeof(int) , hipMemcpyHostToDevice);
        hipMemcpy(d_edge_dst, h_edge_dst, (2*nnz+2708) * sizeof(int) , hipMemcpyHostToDevice);
	
	int hidden_size = 32;	

	if(alloc != hipSuccess) {
    	printf("Feature matrix memcpy failed\n");
	} 
	std::cout << "Dataset captured!\n";
        Data dataset(2708,100,feature_size,label_size,label,h_B);
        free(label);
        free(h_B);
	std::cout << "Dataset captured!\n";
        NeuralNetwork nn(0.02);
        //-----------------------------------------------
        std::cout << "Instance of Neural Network\n";
	nn.addLayer(new NodeAggregator("nodeagg1", d_edge_data, d_row_start, d_edge_dst, 2708, 2*nnz+2708));
        std::cout << "Added Nodeaggregator 1 layer\n";
	nn.addLayer(new LinearLayer("linear1", Shape(feature_size, hidden_size)));
        std::cout << "Added Linear layer 1\n";
	nn.addLayer(new ReLUActivation("relu1"));
        std::cout << "Added relu layer 1\n";
        //-----------------------------------------------
       // nn.addLayer(new NodeAggregator("nodeagg2", d_edge_data, d_row_start, d_edge_dst, 2708, nnz));
       // std::cout << "Added Nodeaggregator layer 2\n";
       // nn.addLayer(new LinearLayer("linear2", Shape(label_size,label_size)));
       // std::cout << "Added Linear layer 2\n";
       // nn.addLayer(new ReLUActivation("relu2"));
       // std::cout << "Added Relu layer 2\n"; 
        //-----------------------------------------------
        nn.addLayer(new NodeAggregator("nodeagg3", d_edge_data, d_row_start, d_edge_dst, 2708, 2*nnz+2708));
        std::cout << "Added Nodeaggregator layer 3\n";
	nn.addLayer(new LinearLayer("linear3", Shape(hidden_size,label_size)));
        std::cout << "Added Linear layer 3\n";
//	nn.addLayer(new ReLUActivation("relu3"));
//        std::cout << "Added Relu layer 3\n"; 
        //-----------------------------------------------
        nn.addLayer(new SoftMax("softmax"));
        std::cout << "Added softmax layer \n";

        std::cout << "Instance of Neural Network complete\n";
	// network training
	Matrix Y;
    int num_train_nodes = 0.6 * (nnodes);
    int num_test_nodes = nnodes - num_train_nodes;

	for (int epoch = 0; epoch < 1000; epoch++) {
		float cost = 0.0;

		Y = nn.forward(dataset.input_features, true);
		nn.backprop(Y,dataset.input_labels,dataset.node_array_device,num_test_nodes);

		cost += bce_cost.cost(Y,dataset.input_labels,dataset.node_array_device, num_test_nodes);
		if (epoch % 10 == 0) {
			std::cout 	<< "Epoch: " << epoch
						<< ", Cost: " << cost / 100
						<< std::endl;
		}
                Y.freeMem();
	}

        float accuracy = 0.0f;
        float final_accuracy = 0.0f;
//	for (int batch = 0; batch < dataset.getNumOfTestBatches(); batch++) {
		Y = nn.forward(dataset.input_features, false);
                Y.allocateHostMemory();
                std::cout << "Y.host allocated:" << Y.host_allocated << "\n";
		Y.copyDeviceToHost();
                std::cout << "Y copied to host "<< "\n";
                accuracy = accuracy + computeAccuracy(Y,dataset.input_labels, dataset.node_array, num_test_nodes);
//	}
        final_accuracy = accuracy;
	// compute accuracy
        
	std::cout << "Accuracy: " << final_accuracy << std::endl;
        hipFree(d_row_start);
        hipFree(d_edge_dst);
        hipFree(d_B);
        hipFree(d_edge_data);
        dataset.input_features.freeMem();
        dataset.input_labels.freeMem();
	return 0;
}

float computeAccuracy(const Matrix& predictions, const Matrix& targets, int *node_array, int num_test_nodes) {
	int correct_predictions = 0;

	for (int i = 0; i < num_test_nodes; i++) { 
        int max_class = 0;
        int max_prediction = -99999;
        for (int j = 0; j < predictions.shape.y; j++) {
            if (predictions[node_array[i] * predictions.shape.y + j] > max_prediction) {
                max_class = j;
                max_prediction = predictions[node_array[i] * predictions.shape.y + j];
            }
        }
        if (targets[node_array[i] * predictions.shape.y + max_class] == 1) {
            correct_predictions++;
        }
	}
	return static_cast<float>(correct_predictions) / (num_test_nodes);
}
