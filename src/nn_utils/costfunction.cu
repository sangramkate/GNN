#include "hip/hip_runtime.h"
#include "costfunction.hh"
#include "nn_exception.hh"

#include <math.h>
#include <iostream>
#include <assert.h>

__global__ void binaryCrossEntropyCost(float* predictions, float* target, int size,int prediction_y, float* cost, int* node_array_device, int num_test_nodes) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
        float partial_cost = 0.0f;
	if (index >= num_test_nodes && index < size) {
            for(int i = 0 ; i < prediction_y; i++){
                int index_train = node_array_device[index];
                partial_cost += (target[index_train* prediction_y + i] * logf(predictions[index_train * prediction_y + i]) 
                                                              + (1.0f - target[index_train * prediction_y + i]) 
                                                              * logf(1.0f - predictions[index_train * prediction_y + i]));
                //partial_cost += target[index* prediction_y + i] * logf(predictions[index * prediction_y + i]); 
                  //partial_cost += 0.5 * (target[index_train* prediction_y + i] - predictions[index_train * prediction_y + i])* (target[index_train* prediction_y + i] - predictions[index_train * prediction_y + i]); 
                  //printf("Partial_cost=%f,log=%f\n",partial_cost,logf(predictions[index_train * prediction_y + i]));
                }
        //printf("Partial_cost=%f,log=%f\n",partial_cost,logf(predictions[index_train * prediction_y + i]);
		  //  dY[index*prediction_y + i] = target[index * prediction_y + i];
        if (partial_cost != logf(0)) {
		    atomicAdd(cost, partial_cost);
        }
	}
}

__global__ void dBinaryCrossEntropyCost(float* predictions, float* target, float* dY, int size,int prediction_y, int* node_array_device, int num_test_nodes) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < num_test_nodes) {
        for (int i = 0; i < prediction_y; i++) {
            dY[node_array_device[index]*prediction_y + i] = 0;
        }
    } else if (index < size) {
            if(index >= num_test_nodes && index < num_test_nodes + 5) {
                for(int i = 0 ; i < prediction_y; i++){ 
                    int index_train = node_array_device[index];
                    dY[index_train*prediction_y + i] = (-target[index_train*prediction_y + i] + predictions[index_train * prediction_y + i] ) / 
                                                        ((1 - predictions[index_train * prediction_y + i]) * predictions[index_train * prediction_y + i]);
		            //dY[node_array_device[index]*prediction_y + i] = target[node_array_device[index] * prediction_y + i]-predictions[node_array_device[index] * prediction_y + i];
                    printf("%f:%f, ",target[node_array_device[index] * prediction_y + i],predictions[node_array_device[index] * prediction_y + i]);
                }
            printf("\n");
           }
	}
}

float CostFunction::cost(Matrix& predictions, Matrix& target, int *node_array_device, int num_test_nodes) {
       // std::cout << "predictions.x:" << predictions.shape.x <<"\n" ;
       // std::cout << "predictions.y:" << predictions.shape.y <<"\n" ;
        //std::cout << "target.x:" << target.shape.x <<"\n" ;
        //std::cout << "target.y:" << target.shape.y <<"\n" ;
	assert(predictions.shape.y == target.shape.y);

	NNException::throwIfDeviceErrorOccurred("Error already happened.");
	float* cost = nullptr;
        hipMalloc(&cost,sizeof(float));
	NNException::throwIfDeviceErrorOccurred("Could not allocate memory.");
        hipMemset(cost, 0.0f, sizeof(float));
	NNException::throwIfDeviceErrorOccurred("Cannot set the data.");
       // std:: cout << "pointer created\n";
       //hipMallocManaged(&cost, sizeof(float));
       // std::cout <<"this gets printed\n";
       //   std:: cout << "Memory Allocated\n";
       //*cost = 0.0f;
       // std:: cout << "cost initialized\n";

	dim3 block_size(256);
      // std:: cout << "dim3 block size\nn";
	dim3 num_of_blocks((predictions.shape.x + block_size.x - 1) / block_size.x);
        //std::cout << "start finding cross entropy\n";
	binaryCrossEntropyCost<<<num_of_blocks, block_size>>>(predictions.data_device, target.data_device,predictions.shape.x,predictions.shape.y, cost, node_array_device, num_test_nodes);
      //  std::cout << "done finding cross entropy\n";
	hipDeviceSynchronize();
	NNException::throwIfDeviceErrorOccurred("Cannot compute binary cross entropy cost.");
        
        float* cost_value = (float*) malloc(sizeof(float));
        hipMemcpy(cost_value,cost,sizeof(float),hipMemcpyDeviceToHost);
        *cost_value = *cost_value * -1 / (predictions.shape.x - num_test_nodes);
	//float cost_value = *cost;
	hipFree(cost);

	return (*cost_value);
}

Matrix& CostFunction::dCost(Matrix& predictions, Matrix& target, Matrix& dY, int *node_array_device, int num_test_nodes) {
	assert(predictions.shape.y == target.shape.y);

	dim3 block_size(256);
	dim3 num_of_blocks((predictions.shape.x + block_size.x - 1) / block_size.x);
	dBinaryCrossEntropyCost<<<num_of_blocks, block_size>>>(predictions.data_device, target.data_device,dY.data_device,predictions.shape.x,predictions.shape.y,node_array_device,num_test_nodes);
	NNException::throwIfDeviceErrorOccurred("Cannot compute derivative for binary cross entropy.");

	return dY;
}
